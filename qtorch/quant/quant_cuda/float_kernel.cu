#include "hip/hip_runtime.h"
#include "quant_kernel.h"
#include "bit_helper.cu"

// quantize a float into a floating point with [exp_bits] exponent and
// [man_bits] mantissa
__global__ void float_kernel_stochastic(float* __restrict__ a,
                                        int* __restrict__ r,
                                        float* o, int size,
                                        int man_bits,
                                        int exp_bits) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  
  unsigned int overflows=0;
  unsigned int underflows=0;
  unsigned int total=size;

  if (index < size) {
    unsigned int rand_prob = (unsigned int) r[index];
    unsigned int target,quantize_bits,quantize_bits_tmp;
    target = FLOAT_TO_BITS(&a[index]);
    float quantized;

    int target_exp = (target << 1 >> 1 >> 23) -127; 
    int min_exp = -((1 << (exp_bits - 1)) - 2);
    bool subnormal = (target_exp < min_exp);
    if (subnormal){
      underflows=underflows+1;
      float shift_float,val;
      int shift_bits = ((127+min_exp)<<23) | (target >> 31 <<31);
      shift_float = BITS_TO_FLOAT(&shift_bits);
      val=a[index]+shift_float;
      target = FLOAT_TO_BITS(&val);
      quantize_bits = round_bitwise_stochastic(target, rand_prob, man_bits);
      quantized = BITS_TO_FLOAT(&quantize_bits) - shift_float;
    }
    else{
      quantize_bits_tmp = round_bitwise_stochastic(target, rand_prob, man_bits);
      quantize_bits = clip_exponent(exp_bits, man_bits, target, quantize_bits_tmp);
      quantized = BITS_TO_FLOAT(&quantize_bits);
      if (quantize_bits!=quantize_bits_tmp){
        overflows=overflows+1;
      }
    }
    o[index] = quantized;
  }
  
  // Log overflows if necessary
  char* QPYTORCH_LOG = getenv("QPYTORCH_LOG");
  if (strcmp(QPYTORCH_LOG,"ALL") == 0){
    char* SLURM_JOB_ID = getenv("SLURM_JOB_ID");
    char file_name[80] = "QPYTORCH_LOG_";
    if (SLURM_JOB_ID!=NULL)
      strcat(file_name, SLURM_JOB_ID);
    strcat(file_name, ".txt");
    FILE *f;
    f = fopen(file_name, "a");
    fprintf(f, "%d %d %d\n",total,overflows,underflows);
    fclose(f);
  }

}

// quantize a float into a floating point with [exp_bits] exponent and
// [man_bits] mantissa
__global__ void float_kernel_nearest(float* __restrict__ a,
                                     float* o, int size,
                                     int man_bits,
                                     int exp_bits) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  unsigned int overflows=0;
  unsigned int underflows=0;
  unsigned int total=size;

  if (index < size) {
    unsigned int target,quantize_bits,quantize_bits_tmp;
    target = FLOAT_TO_BITS(&a[index]);
    float quantized;

    int target_exp = (target << 1 >> 1 >> 23) -127; 
    int min_exp = -((1 << (exp_bits - 1)) - 2);
    bool subnormal = (target_exp < min_exp);
    if (subnormal){
      underflows=underflows+1;
      float shift_float,val;
      int shift_bits = ((127+min_exp)<<23) | (target >> 31 <<31);
      shift_float = BITS_TO_FLOAT(&shift_bits);
      val=a[index]+shift_float;
      target = FLOAT_TO_BITS(&val);
      quantize_bits = round_bitwise_nearest(target, man_bits);
      quantized = BITS_TO_FLOAT(&quantize_bits) - shift_float;
    }
    else{
      quantize_bits_tmp = round_bitwise_nearest(target, man_bits);
      quantize_bits = clip_exponent(exp_bits, man_bits, target, quantize_bits_tmp);
      quantized = BITS_TO_FLOAT(&quantize_bits);
      if (quantize_bits!=quantize_bits_tmp){
        overflows=overflows+1;
      }
    }
    o[index] = quantized;
  }

  // Log overflows if necessary
  char* QPYTORCH_LOG = getenv("QPYTORCH_LOG");
  if (strcmp(QPYTORCH_LOG,"ALL") == 0){
    char* SLURM_JOB_ID = getenv("SLURM_JOB_ID");
    char file_name[80] = "QPYTORCH_LOG_";
    if (SLURM_JOB_ID!=NULL)
      strcat(file_name, SLURM_JOB_ID);
    strcat(file_name, ".txt");
    FILE *f;
    f = fopen(file_name, "a");
    fprintf(f, "%d %d %d\n",total,overflows,underflows);
    fclose(f);
  }

}
